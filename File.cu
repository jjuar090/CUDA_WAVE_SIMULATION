#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

__global__ void test(int* a, int* b, int n) {
    int i = threadIdx.x;

    if (i < n) {
        a[i] += b[i];
    }
}
__global__ void MatrixMultiply(int *a, int *b, int n) {



}
/*
int main() {
    int n = 2;
    int* a;
    int* b;

    // Allocate unified memory for arrays a and b
    hipError_t err = hipMallocManaged(&a, n * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for a: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    err = hipMallocManaged(&b, n * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "hipMallocManaged failed for b: " << hipGetErrorString(err) << std::endl;
        hipFree(a);
        return -1;
    }

    // Initialize arrays
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i + 1;
    }

    // Launch kernel with enough threads to cover all elements
    test << <1, n >> > (a, b, n);

    // Wait for GPU to finish before accessing on host
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "hipDeviceSynchronize failed: " << hipGetErrorString(err) << std::endl;
        hipFree(a);
        hipFree(b);
        return -1;
    }

    // Print results
    for (int i = 0; i < n; i++) {
        std::cout << a[i] << std::endl;
    }

    // Free allocated memory
    hipFree(a);
    hipFree(b);

    return 0;
}
/