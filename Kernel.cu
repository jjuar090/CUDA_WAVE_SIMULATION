#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
// Preallocated memory for result

__global__ void dot(int a_x, int a_y, int b_x, int b_y, int* result) {
    *result = a_x * b_x + a_y * b_y;
}
__global__ void add(int a, int b, int* result) {
    *result = a + b;
}

int dotProduct(int a_x, int a_y, int b_x, int b_y) {
    int result; 
    int* d_result = nullptr;

    hipMallocManaged(&d_result, 4);

    dot << <1, 256 >> > (a_x, a_y, b_x, b_y, d_result);
    //hipDeviceSynchronize();

    hipMemcpy(&result, d_result, 4, hipMemcpyDeviceToHost);

    //hipFree(d_result); // Free memory immediately after use
    hipFree(d_result);
    return result;
    //hipFree(&result);
}


int addNum(int a, int b) {
    int result;

    int* d_result = nullptr;

    hipMallocManaged(&d_result, 4);

    add << <1, 256 >> > (a,b, d_result);

    hipMemcpy(&result, d_result, 4, hipMemcpyDeviceToHost);

    //hipDeviceSynchronize();

    hipFree(d_result);

    return result;
    //hipFree(&result);

}
